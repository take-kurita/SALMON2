
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define ARRAY_INDEX_1D(C_i, Fort_i_start) ((C_i) - (Fort_i_start))
#define ARRAY_INDEX_3D(C_i, C_j, C_k, Fort_i_start, Fort_i_end, Fort_j_start, Fort_j_end, Fort_k_start, Fort_k_end) \
	(((C_i) - (Fort_i_start)) \
	 + ((C_j) - (Fort_j_start)) * ((Fort_i_end) - (Fort_i_start) + 1) \
	 + ((C_k) - (Fort_k_start)) * ((Fort_i_end) - (Fort_i_start) + 1) * ((Fort_j_end) - (Fort_j_start) + 1))
#define ARRAY_INDEX_7D(C0, C1, C2, C3, C4, C5, C6, F0s, F0e, F1s, F1e, F2s, F2e, F3s, F3e, F4s, F4e, F5s, F5e, F6s, F6e) \
	(((C0) - (F0s)) \
	+ ((C1) - (F1s)) * ((F0e) - (F0s) + 1) \
	+ ((C2) - (F2s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1) \
	+ ((C3) - (F3s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) \
	+ ((C4) - (F4s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) * ((F3e) - (F3s) + 1) \
	+ ((C5) - (F5s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) * ((F3e) - (F3s) + 1) * ((F4e) - (F4s) + 1) \
	+ ((C6) - (F6s)) * ((F0e) - (F0s) + 1) * ((F1e) - (F1s) + 1)  * ((F2e) - (F2s) + 1) * ((F3e) - (F3s) + 1) * ((F4e) - (F4s) + 1) * ((F5e) - (F5s) + 1) \
	 )

extern "C" {
__host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex& a, const hipDoubleComplex& b) {
	return make_double2(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__host__ __device__ hipDoubleComplex operator*=(hipDoubleComplex& a, const double b) {
	return a = make_double2(a.x * b, a.y * b);
}

__host__ __device__ hipDoubleComplex operator+=(hipDoubleComplex& a, const hipDoubleComplex& b) {
	return a = make_double2(a.x + b.x, a.y + b.y);
}

// Kernel function for (src/common/nonlocal_potential.f90: l.271)
// Num threads = (im_e - im_s + 1) * (ik_e - ik_s + 1) * (io_e - io_s + 1) * Nlma.
__global__ void zpseudo_kernel(
		// Output & Input
		hipDoubleComplex* const htpsi_zwf,
		// Shape :  (psi%zwf(mg%is_array(1):mg%ie_array(1),  &
		//           mg%is_array(2):mg%ie_array(2),  &
		//           mg%is_array(3):mg%ie_array(3),  &
		//           nspin,info%io_s:info%io_e,info%ik_s:info%ik_e,info%im_s:info%im_e))
		//
		// Input
		const int im_s,
		const int im_e,
		const int ik_s,
		const int ik_e,
		const int io_s,
		const int io_e,
		const int Nspin,
		const int Nlma,
		const int ppg_nps,
		const int natom,
		const int mg_is_array_1,
		const int mg_ie_array_1,
		const int mg_is_array_2,
		const int mg_ie_array_2,
		const int mg_is_array_3,
		const int mg_ie_array_3,
		const int* const ppg_ia_tbl,
		// Shape :  (ppg%ia_tbl(n*natom))
		const int* const ppg_mps,
		// Shape :  (ppg%mps(natom))
		const int* const ppg_jxyz,
		// Shape :  (ppg%jxyz(3,ppg%nps,natom))
		const hipDoubleComplex* const ppg_zekr_uV,
		// Shape :  (ppg%zekr_uV(ppg%nps,ppg%nlma,ik_s:ik_e))
		const double* const ppg_rinv_uvu,
		// Shape :  (ppg%rinv_uvu(n*natom))
		const hipDoubleComplex* const tpsi_zwf
		// Shape :  The same with htpsi_zwf
		) {
	const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;


	const unsigned im_size = im_e - im_s + 1;
	const unsigned ik_size = ik_e - ik_s + 1;
	const unsigned io_size = io_e - io_s + 1;
	const unsigned array_length = im_size * ik_size * io_size * Nspin;
	if (tid >= array_length) {
		return;
	}

	const unsigned im = tid % im_size + im_s;
	const unsigned ik = (tid / im_size) % ik_size + ik_s;
	const unsigned io = (tid / (im_size * ik_size)) % io_size + io_s;
	const unsigned ispin = (tid / (im_size * ik_size * io_size)) + 1;

	for (unsigned ilma = 1; ilma <= Nlma; ilma++) {
		const unsigned ia = ppg_ia_tbl[ARRAY_INDEX_1D(ilma, 1)];
		hipDoubleComplex uVpsi = make_double2(0., 0.);

		for (unsigned j = 1; j <= ppg_mps[ARRAY_INDEX_1D(ia, 1)]; j++) {
			const hipDoubleComplex ppg_zekr_uV_v = ppg_zekr_uV[ARRAY_INDEX_3D(j, ilma, ik, 1, ppg_nps, 1, Nlma, ik_s, ik_e)];
			// calculate conj
			const hipDoubleComplex conjg_ppg_zekr_uV = make_double2(ppg_zekr_uV_v.x, -ppg_zekr_uV_v.y);

			const unsigned ix = ppg_jxyz[ARRAY_INDEX_3D(1, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iy = ppg_jxyz[ARRAY_INDEX_3D(2, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iz = ppg_jxyz[ARRAY_INDEX_3D(3, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			uVpsi += conjg_ppg_zekr_uV * tpsi_zwf[ARRAY_INDEX_7D(
					ix, iy, iz, ispin, io, ik, im,
					mg_is_array_1, mg_ie_array_1,
					mg_is_array_2, mg_ie_array_2,
					mg_is_array_3, mg_ie_array_3,
					1, Nspin,
					io_s, io_e,
					ik_s, ik_e,
					im_s, im_e
					)];
		}

		uVpsi *= ppg_rinv_uvu[ARRAY_INDEX_1D(ilma, 1)];

		for (unsigned j = 1; j <= ppg_mps[ARRAY_INDEX_1D(ia, 1)]; j++) {
			const hipDoubleComplex wrk = uVpsi * ppg_zekr_uV[ARRAY_INDEX_3D(j, ilma, ik, 1, ppg_nps, 1, Nlma, ik_s, ik_e)];

			const unsigned ix = ppg_jxyz[ARRAY_INDEX_3D(1, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iy = ppg_jxyz[ARRAY_INDEX_3D(2, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];
			const unsigned iz = ppg_jxyz[ARRAY_INDEX_3D(3, j, ia, 1, 3, 1, ppg_nps, 1, natoms)];

			const unsigned mem_offset = ARRAY_INDEX_7D(
					ix, iy, iz, ispin, io, ik, im,
					mg_is_array_1, mg_ie_array_1,
					mg_is_array_2, mg_ie_array_2,
					mg_is_array_3, mg_ie_array_3,
					1, Nspin,
					io_s, io_e,
					ik_s, ik_e,
					im_s, im_e
					);
			atomicAdd(&(htpsi_zwf[mem_offset].x), wrk.x);
			atomicAdd(&(htpsi_zwf[mem_offset].y), wrk.y);
		}
	}
}

void zpseudo_cuda(
		// Output & Input
		hipDoubleComplex* const htpsi_zwf,
		// Input
		const int n,
		const int im_s,
		const int im_e,
		const int ik_s,
		const int ik_e,
		const int io_s,
		const int io_e,
		const int Nspin,
		const int Nlma,
		const int ppg_nps,
		const int natom,
		const int mg_is_array_1,
		const int mg_ie_array_1,
		const int mg_is_array_2,
		const int mg_ie_array_2,
		const int mg_is_array_3,
		const int mg_ie_array_3,
		const int* const ppg_ia_tbl,
		const int* const ppg_mps,
		const int* const ppg_jxyz,
		const hipDoubleComplex* const ppg_zekr_uV,
		const double* const ppg_rinv_uvu,
		hipDoubleComplex* const tpsi_zwf
		) {
	const unsigned im_size = im_e - im_s + 1;
	const unsigned ik_size = ik_e - ik_s + 1;
	const unsigned io_size = io_e - io_s + 1;
	const unsigned num_threads = im_size * ik_size * io_size * Nspin;

	const unsigned block_size = 256;
	const unsigned grid_size = (num_threads + block_size - 1) / block_size;

	zpseudo_kernel<<<grid_size, block_size>>>(
		htpsi_zwf,
		// Input
		im_s,
		im_e,
		ik_s,
		ik_e,
		io_s,
		io_e,
		Nspin,
		Nlma,
		ppg_nps,
		natom,
		mg_is_array_1,
		mg_ie_array_1,
		mg_is_array_2,
		mg_ie_array_2,
		mg_is_array_3,
		mg_ie_array_3,
		ppg_ia_tbl,
		ppg_mps,
		ppg_jxyz,
		ppg_zekr_uV,
		ppg_rinv_uvu,
		tpsi_zwf
		);
	hipDeviceSynchronize();
}
} // extern "C"
